﻿
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>


using namespace std;


__global__ void perceptron(const float* W, const float* x, float b, float* out, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		atomicAdd(out, W[idx] * x[idx]); //thread-safe addition.
	}
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    const int N = 1024;
    //float h_W[N] = { 1,2,3,4,5,6,7,8,9,10 };
    //float h_x[N] = { 1,2,3,4,5,6,7,8,9,10 };
    float b = 1.5f;
    float h_out = 0.0f;

    h_out = b;

    float h_W[N], h_x[N];

    for (int i = 0; i < N; i++) {
        h_W[i] = i;
        h_x[i] = i;
    }



    float* d_W, * d_x, * d_out;

    hipMalloc(&d_W, N * sizeof(float));
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_out, sizeof(float));

    hipMemcpy(d_W, h_W, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out, &h_out, sizeof(float), hipMemcpyHostToDevice);


    // launch kernel
    int threads = 256;
    int blocks = (N + threads - 1) / threads; //ceil(blocksize/N)
    int REPEATS = 100;
    float milliseconds = 0;

    hipEventRecord(start);
    for (int i = 0; i < REPEATS; i++) {
        h_out = b;
        hipMemcpy(d_out, &h_out, sizeof(float), hipMemcpyHostToDevice);
        perceptron <<<blocks, threads >>> (d_W, d_x, b, d_out, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);


    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total Elapsed Time: %f ms\n\n\n", milliseconds / REPEATS);



    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);





    cout << "W vector [";

    for (int i = 0; i < N; i++) {
        cout << h_W[i]<<",";
    }
    cout << "]\n\n\n";


    cout << "x vector [";

    for (int i = 0; i < N; i++) {
        cout << h_x[i] << ",";
    }
    cout << "]\n\n\n";



    printf("Perceptron output: %f\n", h_out);

    hipFree(d_W);
    hipFree(d_x);
    hipFree(d_out);

    return 0;
}